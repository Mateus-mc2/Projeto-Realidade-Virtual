#include "hip/hip_runtime.h"
#include "gpu_triangular_object.h"

#include <thrust/complex.h>

#include "gpu_linear_algebra.h"
#include "math_lib.h"

namespace gpu {

GPUTriangularObject::GPUTriangularObject(const GPUMaterial &material,
                                         const GPUVector<float3> &vertices,
                                         const GPUVector<int3> &faces)
    : kEps(1.0e-3f),
      material_(material),
      planes_coeffs_(faces.size()),
      linear_systems_(3 * faces.size()) {
  for (int i = 0; i < faces.size(); ++i) {
    // Get plane equation (coefficients) - we are assuming the .obj files provide us the 
    // correct orientation of the vertices.
    float3 a = vertices[faces[i].x];
    float3 b = vertices[faces[i].y];
    float3 c = vertices[faces[i].z];

    float3 ab = make_float3(b.x - a.x, b.y - a.y, b.z - a.z);
    float3 ac = make_float3(c.x - a.x, c.y - a.y, c.z - a.z);
    float3 normal = math::Cross(ab, ac);
    float4 coeffs;

    coeffs.x = normal.x;
    coeffs.y = normal.y;
    coeffs.z = normal.z;
    coeffs.w = -math::InnerProduct(a, normal);

    this->planes_coeffs_[i] = coeffs;

    // Get system's LUP decomposition matrices.
    GPUMatrix L = GPUMatrix::Identity(3, 3);
    GPUMatrix U(3, 3);

    U(0, 0) = a.x;
    U(1, 0) = a.y;
    U(2, 0) = a.z;

    U(0, 1) = b.x;
    U(1, 1) = b.y;
    U(2, 1) = b.z;

    U(0, 2) = c.x;
    U(1, 2) = c.y;
    U(2, 2) = c.z;

    GPUMatrix P;
    LUPDecomposition(&L, &U, &P);

    this->linear_systems_[3 * i] = L;
    this->linear_systems_[3 * i + 1] = U;
    this->linear_systems_[3 * i + 2] = P;
  }
}

 GPUTriangularObject& GPUTriangularObject::operator=(const GPUTriangularObject &obj) {
  if (this != &obj) {
    this->material_ = obj.material();
    this->planes_coeffs_ = obj.planes_coeffs();
    this->linear_systems_ = obj.linear_systems();
  }

  return *this;
}

__host__ __device__ float GPUTriangularObject::GetIntersectionParameter(const GPURay &ray,
                                                                        float3 *normal) const {
  float min_t = -1.0f;
  auto is_inner_point = [this](float a, float b, float c) -> bool { 
    return math::IsAlmostEqual(a + b + c, 1.0f, this->kEps) && a >= 0 && b >= 0 && c >= 0 && a <= 1
           && b <= 1 && c <= 1;
  };

  bool has_intersection = false;

  // Get nearest intersection point - need to check every single face of the object.
  for (int i = 0; i < this->planes_coeffs_.size(); ++i) {
    const float3 current_normal = make_float3(this->planes_coeffs_[i].x, this->planes_coeffs_[i].y,
                                              this->planes_coeffs_[i].z);
    const float numerator = -(this->planes_coeffs_[i].x * ray.origin.x +
                              this->planes_coeffs_[i].y * ray.origin.y +
                              this->planes_coeffs_[i].z * ray.origin.z +
                              this->planes_coeffs_[i].w);
    const float denominator = (current_normal.x * ray.direction.x +
                               current_normal.y * ray.direction.y +
                               current_normal.z * ray.direction.z);

    // Test if the ray and this plane are parallel (or if this plane contains the ray).
    // Returns a negative (dummy) parameter t if this happens.
    if (math::IsAlmostEqual(denominator, 0.0f, this->kEps)) {
      return -1.0;
    }

    float curr_t = numerator / denominator;
    GPUMatrix intersection_point(3, 1);

    intersection_point(0, 0) = ray.origin.x + curr_t * ray.direction.x;
    intersection_point(1, 0) = ray.origin.y + curr_t * ray.direction.y;
    intersection_point(2, 0) = ray.origin.z + curr_t * ray.direction.z;

    intersection_point = this->linear_systems_[3 * i + 2] * intersection_point;
    GPUMatrix barycentric_coords(3, 1);

    ApplyForwardSubstitution(this->linear_systems_[3 * i], intersection_point, &barycentric_coords);
    ApplyBackSubstitution(this->linear_systems_[3 * i + 1], barycentric_coords, &barycentric_coords);

    bool is_inside = is_inner_point(barycentric_coords(0, 0), barycentric_coords(1, 0),
                                    barycentric_coords(2, 0));

    if (is_inside && (!has_intersection || (min_t > curr_t && curr_t > this->kEps))) {
      min_t = curr_t;
      (*normal) = current_normal;
      has_intersection = true;
    }
  }

  if (has_intersection) {
    math::Normalize(normal);
  }

  return min_t;
}

}  // namespace gpu
